//
// Created by Андрей on 14.08.2023.
//

#include "ImageAdapter.h"

__host__ __device__ void ImageAdapter::setPixelColor(int x, int y, ColorRGB color)
{
	colorMatrix[x * _width + y] = color;
}
__host__  __device__ int ImageAdapter::getWidth()
{
	return _width;
}
__host__  __device__ int ImageAdapter::getHeight()
{
	return _height;
}
__host__  __device__ ImageAdapter ImageAdapter::getImage()
{
	return *this;
}
__host__ __device__  ImageAdapter::ImageAdapter()
{
	_width = 1;
	_height = 1;
	//cpuErrorCheck(hipMalloc((void**)&colorMatrix, 1 * sizeof(ColorRGB)));
}
__host__ __device__  ImageAdapter::ImageAdapter(int width, int height)
{
	_width = width;
	_height = height;
	cpuErrorCheck(hipMalloc((void**)&colorMatrix, _width * _height * sizeof(ColorRGB)));
}
__host__  ImageAdapter::~ImageAdapter()
{
	cpuErrorCheck(hipFree(colorMatrix));
}