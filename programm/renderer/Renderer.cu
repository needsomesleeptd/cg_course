#include "hip/hip_runtime.h"
//
// Created by Андрей on 09.08.2023.
//

#include "Renderer.cuh"
#include "scene.h"

#include "color.h"
#include "../scene/scene.h"
#include "../object/invisibleObject/camera.h"
#include "../image_adapter/ImageAdapter.h"
#include "../object/object.h"

#include "LightSource.cuh"
#include "../utils/cudaUtils.h"
#include "CudaShape.cuh"
#include "../object/invisibleObject/lightSource/LightSource.cuh"
#include "../math_primitives/ray/ray.h"
#include "material.h"
#include "camera.h"

#include <hip/hip_runtime_api.h>

//const ColorRGB backGround = ColorRGB(0, 0, 0);

__device__  ColorRGB rayTrace(const Ray& tracedRay,
	ColorRGB& otherColor,
	int curDepth,
	CudaArray<CudaShape>* objects,
	LightSource* lightSource)
{
	CudaShape* closestShape;
	float t = maxRange;
	ColorRGB finalColor;

	for (int i = 0; i < objects->n; i++)
	{
		//printf("%d \n" ,i);
		float intersection_t = (objects->values[i]).intersection(tracedRay);
		//std::cout << intersection_t << std::endl;
		if (intersection_t > 0 || fabs(intersection_t) < EPS)
		{
			if (intersection_t < t)
			{
				t = intersection_t;
				closestShape = &objects->values[i];
			}
		}
	}
	//0x55b4c14f9288
	//printf("\nobjects = %p\n lightSource = %p\n\n",objects,lightSource);
	//printf("\nobject1 = %p\n object2 = %p\n\n",objects,lightSource);
	if (abs(t - maxRange) < EPS)
	{
		return  ColorRGB(0, 0, 0); //Returning background color
	}
	//printf("light source = %f",lightSource->getColor().R);
	//printf("function = %p\n",lightSource->getColor());
	//lightSource->getColor();
	LightSource* currentLightSource = lightSource;

	VecD3 intersectionPoint = tracedRay.getPoint(t);
	//printf("Position  == %f",currentLightSource->getPosition().x());
	VecD3 lightVector = glm::normalize(intersectionPoint - currentLightSource->getPosition());
	//lightSource->getColor();

	VecD3 shapeNormal = glm::normalize(closestShape->getNormal(intersectionPoint));

	Material shapeMaterial = closestShape->getMaterial();
	float ambientIntensivity = shapeMaterial._k_a * currentLightSource->getIntensivity();
	finalColor = shapeMaterial._color * ambientIntensivity + finalColor;
	float diffuseLight = dot(shapeNormal, lightVector);

	if (shapeMaterial._k_d > 0)
	{
		if (diffuseLight > 0)
		{
			//std::cout << " diffuseLight" << diffuseLight << std::endl;
			ColorRGB diffuseColorRay = currentLightSource->getColor() * diffuseLight * shapeMaterial._k_d;
			finalColor = shapeMaterial._color * diffuseColorRay + finalColor;

		}
	}
	if (shapeMaterial._k_s > 0)
	{

		Ray reflected = tracedRay.calculateReflected(shapeNormal, intersectionPoint);
		float specularDot = dot(reflected.D, tracedRay.D);
		//std::cout << " diffuseLight" << diffuseLight << std::endl;
		if (specularDot > 0.0)
		{
			//float spec = powf( specularDot, 20 ) * shapeMaterial._k_s;
			finalColor = currentLightSource->getColor() * specularDot * shapeMaterial._k_s + finalColor;
		}
	}

	return finalColor;
	/*if (shapeMaterial._k_s > 0.0f)
	{
		VecD3 N = closestShape->getNormal(intersectionPoint);
		Ray reflected = tracedRay.calculateReflected(shapeNormal, intersectionPoint);
		if (curDepth < maxDepth)
		{
			ColorRGB rcol(0, 0, 0);
			rayTrace(reflected, rcol, scene, curDepth + 1, objects, lightSource);
			finalColor = rcol * shapeMaterial._k_s * closestShape->getMaterial()._color + finalColor;
		}
	}*/
}

__device__  Ray createRay(int x, int y, Camera* currentCamera, ImageAdapter* image)
{
	float imageHeight = image->_height; //image->getHeight();
	float imageWidth = image->_width; //image->getWidth();
	VecD3 viewPoint = currentCamera->getViewPoint();
	VecD3 l = viewPoint - float(imageWidth / 2);
	VecD3 r = viewPoint + float(imageWidth / 2);

	VecD3 up = viewPoint - float(imageHeight / 2);
	VecD3 down = viewPoint + float(imageHeight / 2);

	//VecD3 u_deformation = float(x) * (r - l) / float(imageWidth);
	//VecD3 v_deformation = float(y) * (up - down) / float(imageHeight);//TODO::fix ray origin
	/*VecD3 ray_origin = viewPoint + u_deformation * VecD3(1, 0, 0) + v_deformation * VecD3(0, 1, 0);
	return Ray(ray_origin, -currentCamera->getViewDirection());*/
	glm::vec2 coord = { (float)x / (float)imageWidth, (float)y / (float)imageWidth };
	coord = coord * 2.0f - 1.0f; // -1 -> 1

	VecD4 target = currentCamera->getInverseProjectionMatrix() * VecD4(coord.x, coord.y, 1, 1);
	VecD3 rayDirection = VecD3(currentCamera->getInverseViewMatrix() * VecD4(glm::normalize(VecD3(target) / target.w), 0)); // World space
	return Ray(viewPoint, rayDirection);
	/*VecD3 dir =VecD3(4*x,3*y,0) - viewPoint;
	return Ray(viewPoint,dir);*/
}

__device__ ColorRGB renderPixel(int x,
	int y,
	Camera* camera,
	CudaArray<CudaShape>* objects,
	LightSource* lightSource,
	ImageAdapter* image)
{
	Ray tracedRay = createRay(x, y, camera, image);
	ColorRGB finalColor;
	finalColor = rayTrace(tracedRay, finalColor, 0, objects, lightSource);
	return finalColor;
}

__global__ void renderSceneCuda(Camera** camera,
	CudaArray<CudaShape>* objects,
	LightSource** lightSource,
	ImageAdapter* image)
{

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	//printf("height = %d width = %d\n",image->_height,image->_width);
	//printf("%d %d\n",i,j);
	if (i >= image->_width || j >= image->_height)
		return;
	ColorRGB pixelColor = renderPixel(i, j, *camera, objects, *lightSource, image);
	pixelColor.normalize();
	//std::cout << pixelColor.R <<" "<< pixelColor.G << " "<< pixelColor.B << std::endl;

	image->setPixelColor(i, j, pixelColor);
	//printf("%p",image->colorMatrix);
	//pixelColor.R = 1;
}


__global__ void createLightSource(
	LightSource** lightSource,VecD3 position,float intensivity)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		(*lightSource) = new LightSource(position,intensivity);
	}
}

__global__ void destroyLightSource(
	LightSource** lightSource)
{
	delete *lightSource;
}


__global__ void createCamera(
	Camera** camera,VecD3 coordinates,VecD3 direction)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		(*camera) = new Camera(coordinates,direction);
	}
}



__host__ ImageAdapter* Renderer::renderScene(std::shared_ptr<Scene> scene)
{
	int blockX = 5;
	int blockY = 5;
	int nx = 600;
	int ny = 600;
	ImageAdapter hostImage;
	hostImage._width = nx;
	hostImage._height = ny;
	ImageAdapter* deviceImage;

	cpuErrorCheck(hipMalloc((void**)&(hostImage.colorMatrix), sizeof(ColorRGB) * nx * ny));
	cpuErrorCheck(hipMalloc(&deviceImage, sizeof(ImageAdapter)));

	cpuErrorCheck(hipMemcpy(deviceImage, &hostImage, sizeof(ImageAdapter), hipMemcpyHostToDevice));


	std::shared_ptr<Camera> cameraHost = scene->getCamera();



	std::shared_ptr<LightSource> lightSourceHost = std::dynamic_pointer_cast<LightSource>(scene->getLightSource());


	//creating lightSource
	LightSource** lightSourceDevice;
	cpuErrorCheck(hipMalloc((void **)&lightSourceDevice, sizeof(LightSource**)));
	createLightSource<<<1,1>>>(lightSourceDevice,VecD3(1,1,1),1.0f);
	cpuErrorCheck(hipGetLastError());
	cpuErrorCheck(hipDeviceSynchronize());

	//creating Camera
	Camera** cameraDevice;
	cpuErrorCheck(hipMalloc((void **)&cameraDevice, sizeof(Camera**))); //TODO::make camera class transparent
	createCamera<<<1,1>>>(cameraDevice,cameraHost->_cameraStructure->getCoordinates(),cameraHost->_cameraStructure->getViewDirection());
	cpuErrorCheck(hipGetLastError());
	cpuErrorCheck(hipDeviceSynchronize());



	/*cpuErrorCheck(hipMalloc((void**)&(lightSourceDevice), sizeof(LightSource)));
	cpuErrorCheck(hipMemcpy(lightSourceDevice, lightSourceHost.get() ,sizeof(LightSource), hipMemcpyHostToDevice));*/


	std::vector<std::shared_ptr<BaseObject>> hostObjects = scene->getModels();

	CudaArray<CudaShape> hostVector;
	hostVector.n = hostObjects.size();
	hostVector.values = (CudaShape*)malloc(sizeof(CudaShape) * hostVector.n);

	for (int i = 0; i < hostObjects.size(); i++)
	{
		std::shared_ptr<BaseShape> hostShape = std::dynamic_pointer_cast<BaseShape>(hostObjects[i]);

		/*void* deviceShape;
		switch (hostShape.getShapeType())
		{
			case CudaShapeType::sphere:
				cpuErrorCheck(hipMalloc((void**)&(deviceShape), sizeof(Sphere)));
				cpuErrorCheck(hipMemcpy(deviceShape, hostShape.get() sizeof(Sphere), hipMemcpyHostToDevice));
			break;
		}
		CudaShape *cudaDeviceShape;
		cpuErrorCheck(hipMalloc((void**)(CudaDevice), sizeof(CudaArray)));*/
		CudaShape hostCudaShape(CudaShapeType::sphere,hostShape.get());
		hostVector.values[i] = hostCudaShape;
	}
	CudaArray<CudaShape> transferArray;
	transferArray.n = hostObjects.size();
	cpuErrorCheck(hipMalloc((void**)&(transferArray.values), sizeof(CudaShape) * transferArray.n));
	cpuErrorCheck(hipMemcpy(transferArray.values, hostVector.values ,sizeof(CudaShape) * transferArray.n, hipMemcpyHostToDevice));

	CudaArray<CudaShape>* deviceVector;
	cpuErrorCheck(hipMalloc((void**)&(deviceVector), sizeof(CudaArray<CudaShape>)));
	cpuErrorCheck(hipMemcpy(deviceVector, &transferArray ,sizeof(CudaArray<CudaShape>), hipMemcpyHostToDevice));




	dim3 blocks(nx / blockX, ny / blockY);
	dim3 threads(blockX, blockY);
	renderSceneCuda<<<blocks, threads>>>(cameraDevice, deviceVector, lightSourceDevice, deviceImage);
	cpuErrorCheck(hipGetLastError());
	cpuErrorCheck(hipDeviceSynchronize());

	ImageAdapter* resultImage;
	resultImage = (ImageAdapter*)malloc(sizeof(ImageAdapter)); //Forced to allocate on heap because of destructor
	//TODO::Create normal destructor for image
	hipMemcpy(resultImage, deviceImage, sizeof(ImageAdapter), hipMemcpyDeviceToHost);
	void* deviceColorMap = resultImage->colorMatrix;
	resultImage->colorMatrix = (ColorRGB*)malloc(sizeof(ColorRGB) * nx * ny);
	hipMemcpy(resultImage->colorMatrix, deviceColorMap, sizeof(ColorRGB) * nx * ny, hipMemcpyDeviceToHost);



	hipFree(hostImage.colorMatrix);
	hipFree(deviceImage);
	hipFree(transferArray.values);
	hipFree(deviceVector);

	destroyLightSource<<<1, 1>>>(lightSourceDevice);
	hipFree(lightSourceDevice);

	resultImage->_width = nx;
	resultImage->_height = ny;
	return resultImage;
}

/*Renderer::Renderer(QGraphicsScene* scene)
{
	_scene = scene;
}*/

__host__   void Renderer::getImage(ImageAdapter* image)
{
	;
}
__device__ Ray Renderer::createRay(int x, int y, Camera* currentCamera)
{
	//TODO:: make this
}
__device__ ColorRGB Renderer::renderPixel(int x, int y, Scene* scene, Camera* camera)
{
	//TODO:: make this
};
__device__ void Renderer::rayTrace(const Ray& tracedRay, ColorRGB& finalColor, Scene* scene, int curDepth)
{
	//TODO:: make this
};
